#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#ifndef RAND_MAX
#define RAND_MAX 32767
#endif

#include "neuralNetwork.h"

double init_weight(){ return ((double) rand())/ ((double)RAND_MAX);}


void shuffle(int* array, size_t n){
	if(n > 1){
		size_t i;
		for(i = 0; i < n - 1; i++){
			size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
			int t = array[j];
			array[j] = array[i];
			array[i] = t;
		}
	}

}


int main(int argc, char** argv){
	if(argc < 2){
		printf("add some params");
		exit(0);
	}

	//int gridSize = 4;
	//int blockSize = 1;
	time_t t;
 	srand((unsigned)time(&t));
	// Set the learning rate & epochs
	int epochs = 10000;
	double lr = 1.0f;

	int numInputs = 2;
	int numHiddenNodes = 4;
	int numOutputs = 1;

	double training_inputs[8] = {0.0f,0.0f, 0.0f, 1.0f, 1.0f, 0.0f, 1.0f, 1.0f};
	double training_outputs[4] = {0.0f, 1.0f, 1.0f, 0.0f};
	int trainingSetOrder[] = {0,1,2,3};

	// Initialize all the arrays into memory
	double* hiddenLayer = (double*) malloc(numHiddenNodes * sizeof(double));
	double* outputLayer = (double*) malloc(numOutputs * sizeof(double));

	double* hiddenLayerBias = (double*) malloc(numHiddenNodes * sizeof(double));
	double* outputLayerBias = (double*) malloc(numOutputs * sizeof(double));

	double* hiddenWeights = (double*)malloc(numInputs * numHiddenNodes* sizeof(double));
	double* outputWeights = (double*)malloc(numHiddenNodes * numOutputs * sizeof(double));

	//cuda
	double* cuHiddenLayer;
	double* cuOutputLayer;
	double* cuHiddenLayerBias;
	double* cuOutputLayerBias;
	double* cuOutputWeights;
	double* cuHiddenWeights;
	double* cuTrainingInputs;
	double* cuTrainingOutputs;
	int* cuTrainingSetOrder;

	hipMalloc((void**)&cuHiddenLayer, numHiddenNodes * sizeof(double));
	hipMalloc((void**)&cuOutputLayer, numOutputs * sizeof(double));
	hipMalloc((void**)&cuHiddenLayerBias, numHiddenNodes * sizeof(double));
	hipMalloc((void**)&cuOutputLayerBias, numOutputs * sizeof(double));
	hipMalloc((void**)&cuHiddenWeights, numInputs * numHiddenNodes * sizeof(double));
	hipMalloc((void**)&cuTrainingInputs, 8 * sizeof(double));
	hipMalloc((void**)&cuTrainingOutputs, 4 * sizeof(double));
	hipMalloc((void**)&cuTrainingSetOrder, 4 * sizeof(int));
	hipMalloc((void**)&cuOutputWeights, numHiddenNodes * numOutputs *  sizeof(double));

	// Initialize All The Weights
	for(int i = 0; i < numInputs; i++){
		for(int j = 0; j < numHiddenNodes; j++){
			hiddenWeights[(i * 2) + j] = init_weight();
		}
	}
	for(int i=0;i<numHiddenNodes;i++){
		hiddenLayerBias[i] = init_weight();
		for(int j=0; j<numOutputs; j++){
			outputWeights[(2 * i )+ j] = init_weight();
		}
	}

	for(int i = 0; i<numOutputs; i++){
		outputLayerBias[i] = init_weight();
	}


	int numTrainingSets = 4; 
	hipMemcpy(cuHiddenLayer, hiddenLayer, numHiddenNodes * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputLayer, outputLayer, numOutputs * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuHiddenLayerBias, hiddenLayerBias, numHiddenNodes * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputLayerBias, outputLayerBias, numOutputs * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuHiddenWeights, hiddenWeights, numInputs * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuTrainingInputs, training_inputs, 8 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuTrainingOutputs, training_outputs, 4 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuTrainingSetOrder, trainingSetOrder, 4 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputWeights, outputWeights, numHiddenNodes * sizeof(double), hipMemcpyHostToDevice);









	for(int n = 0; n < epochs; n++){
		
		shuffle(trainingSetOrder, numTrainingSets);
		
		for(int x = 0; x < numTrainingSets; x++){
			int i = trainingSetOrder[x]; 
			//printf("%d\n", n);
			forwardFeed<<<1, 1>>>(cuTrainingInputs, cuHiddenWeights, cuHiddenLayer, cuOutputLayer, cuOutputWeights, cuOutputLayerBias, cuHiddenLayerBias, numHiddenNodes, numInputs, numOutputs, i);	
			hipDeviceSynchronize(); 

			backpropogate<<<1, 1>>>(cuTrainingInputs, cuHiddenLayer, cuHiddenWeights, cuOutputLayer, cuOutputWeights, cuTrainingOutputs, cuHiddenLayerBias, cuOutputLayerBias, numHiddenNodes, numInputs, numOutputs, i, lr);	

			hipDeviceSynchronize();
			// Predict Function
				
		}

			
	
	}
	
	
	//  Transfer memory off of GPU to CPU 
	// To run predict we have to:
       	// Specify user inputs	

       // Create two pieces of test input
       double test_input[2] ={atof(argv[1]), atof(argv[2])}; 
       
       double* cuInputs; 
       
       hipMalloc((void**)&cuInputs, 2 * sizeof(double));
       
       hipMemcpy(cuInputs, test_input, 2 * sizeof(double), hipMemcpyHostToDevice);
       
       forwardFeed<<<1, 1>>>(cuInputs, cuHiddenWeights, cuHiddenLayer, cuOutputLayer, cuOutputWeights, cuOutputLayerBias, cuHiddenLayerBias, numHiddenNodes, numInputs, numOutputs, 0);	
       hipDeviceSynchronize();

       // Transfer the memory off of the GPU to the CPU 
       //outputLayer[0] = 15.00f;
       hipMemcpy(outputLayer, cuOutputLayer, numOutputs * sizeof(double), hipMemcpyDeviceToHost);

       // Run the forward feed function given the Hidden Weights

       // Then we output 

       printf("%f\n", outputLayer[0]);	


}


