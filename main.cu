#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#ifndef RAND_MAX
#define RAND_MAX 32767
#endif

#include "neuralNetwork.h"

float init_weight(){ return ((float) rand())/ ((float)RAND_MAX);}


void shuffle(int* array, size_t n){
	if(n > 1){
		size_t i;
		for(i = 0; i < n - 1; i++){
			size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
			int t = array[j];
			array[j] = array[i];
			array[i] = t;
		}
	}

}


int main(int argc, char** argv){
	if(argc < 2){
		printf("add some params");
		exit(0);
	}

	//int gridSize = 4;
	//int blockSize = 1;
	time_t t;
 	srand((unsigned)time(&t));
	// Set the learning rate & epochs
	int epochs = 10000;
	float lr = 1.0f;

	int numInputs = 2;
	int numHiddenNodes = 4;
	int numOutputs = 1;

	float training_inputs[8] = {0.0f,0.0f, 0.0f, 1.0f, 1.0f, 0.0f, 1.0f, 1.0f};
	float training_outputs[4] = {0.0f, 1.0f, 1.0f, 0.0f};
	int trainingSetOrder[] = {0,1,2,3};
	int numTrainingSets = 4;

	// Initialize all the arrays into memory
	float* hiddenLayer = (float*) malloc(numHiddenNodes * sizeof(float));
	float* outputLayer = (float*) malloc(numOutputs * sizeof(float));
	
	float* hiddenLayerAct = (float*) malloc(numHiddenNodes * sizeof(float));
	float* outputLayerAct = (float*) malloc(numOutputs * sizeof(float));

	float* hiddenLayerBias = (float*) malloc(numHiddenNodes * sizeof(float));
	float* outputLayerBias = (float*) malloc(numOutputs * sizeof(float));

	float* hiddenWeights = (float*)malloc(numInputs * numHiddenNodes* sizeof(float));
	float* outputWeights = (float*)malloc(numHiddenNodes * numOutputs * sizeof(float));

	//cuda
	float* cuHiddenLayer;
	float* cuOutputLayer;
	
	float* cuHiddenLayerAct;
	float* cuOutputLayerAct;
	
	float* cuHiddenLayerBias;
	float* cuOutputLayerBias;
	float* cuOutputWeights;
	float* cuHiddenWeights;
	float* cuTrainingInputs;
	float* cuTrainingOutputs;
	int* cuTrainingSetOrder;

	hipMalloc((void**)&cuHiddenLayer, numHiddenNodes * sizeof(float));
	hipMalloc((void**)&cuOutputLayer, numOutputs * sizeof(float));
	
	hipMalloc((void**)&cuHiddenLayerAct, numHiddenNodes * sizeof(float));
	hipMalloc((void**)&cuOutputLayerAct, numOutputs * sizeof(float));
	
	hipMalloc((void**)&cuHiddenLayerBias, numHiddenNodes * sizeof(float));
	hipMalloc((void**)&cuOutputLayerBias, numOutputs * sizeof(float));
	hipMalloc((void**)&cuHiddenWeights, numInputs * numHiddenNodes * sizeof(float));
	hipMalloc((void**)&cuTrainingInputs, 8 * sizeof(float));
	hipMalloc((void**)&cuTrainingOutputs, 4 * sizeof(float));
	hipMalloc((void**)&cuTrainingSetOrder, 4 * sizeof(int));
	hipMalloc((void**)&cuOutputWeights, numHiddenNodes * numOutputs *  sizeof(float));

	// Initialize All The Weights
	for(int i = 0; i < numInputs; i++){
		for(int j = 0; j < numHiddenNodes; j++){
			hiddenWeights[(i * 2) + j] = init_weight();
		}
	}
	for(int i=0;i<numHiddenNodes;i++){
		hiddenLayerBias[i] = init_weight();
		for(int j=0; j<numOutputs; j++){
			outputWeights[(2 * i )+ j] = init_weight();
		}
	}

	for(int i = 0; i<numOutputs; i++){
		outputLayerBias[i] = init_weight();
	}


	//cuda memory copy to device
	hipMemcpy(cuHiddenLayer, hiddenLayer, numHiddenNodes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputLayer, outputLayer, numOutputs * sizeof(float), hipMemcpyHostToDevice);
	
	hipMemcpy(cuHiddenLayerAct, hiddenLayerAct, numHiddenNodes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputLayerAct, outputLayerAct, numOutputs * sizeof(float), hipMemcpyHostToDevice);

	
	hipMemcpy(cuHiddenLayerBias, hiddenLayerBias, numHiddenNodes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputLayerBias, outputLayerBias, numOutputs * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuHiddenWeights, hiddenWeights, numInputs * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuTrainingInputs, training_inputs, 8 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuTrainingOutputs, training_outputs, 4 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cuTrainingSetOrder, trainingSetOrder, 4 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuOutputWeights, outputWeights, numHiddenNodes * sizeof(float), hipMemcpyHostToDevice);



//------------------------------------------------------------------------------------------------------------------
//start epochs

	for(int n = 0; n < epochs; n++){
		shuffle(trainingSetOrder, numTrainingSets);
		for(int x = 0; x < numTrainingSets; x++){
			int i = trainingSetOrder[x];
			forwardFeedP1<<<1, 1>>>(cuTrainingInputs, cuHiddenWeights, cuHiddenLayer, cuOutputLayer, cuOutputWeights, cuOutputLayerBias, cuHiddenLayerBias, numHiddenNodes, numInputs, numOutputs, i, cuHiddenLayerAct);
			hipDeviceSynchronize();

			sigmoidActivationForward<<<1,4>>>(cuHiddenLayerAct,cuHiddenLayer,4,1);
			hipDeviceSynchronize();

			forwardFeedP2<<<1, 1>>>(cuTrainingInputs, cuHiddenWeights, cuHiddenLayer, cuOutputLayer, cuOutputWeights, cuOutputLayerBias, cuHiddenLayerBias, numHiddenNodes, numInputs, numOutputs, i, cuOutputLayerAct);
			hipDeviceSynchronize();
			
			sigmoidActivationForward<<<1,1>>>(cuOutputLayerAct,cuOutputLayer,1,1);
			hipDeviceSynchronize();

			backpropogate<<<1, 1>>>(cuTrainingInputs, cuHiddenLayer, cuHiddenWeights, cuOutputLayer, cuOutputWeights, cuTrainingOutputs, cuHiddenLayerBias, cuOutputLayerBias, numHiddenNodes, numInputs, numOutputs, i, lr);
			hipDeviceSynchronize();
		}
	}
	
	// Predict Function

	// Create two pieces of test input
	float test_input[2] ={atof(argv[1]), atof(argv[2])};
	float* cuInputs;
	hipMalloc((void**)&cuInputs, 2 * sizeof(float));
	hipMemcpy(cuInputs, test_input, 2 * sizeof(float), hipMemcpyHostToDevice);
        
        forwardFeedP1<<<1, 1>>>(cuInputs, cuHiddenWeights, cuHiddenLayer, cuOutputLayer, cuOutputWeights, cuOutputLayerBias, cuHiddenLayerBias, numHiddenNodes, numInputs, numOutputs, 0,cuHiddenLayerAct);
			hipDeviceSynchronize();

	forwardFeedP2<<<1, 1>>>(cuInputs, cuHiddenWeights, cuHiddenLayer, cuOutputLayer, cuOutputWeights, cuOutputLayerBias, cuHiddenLayerBias, numHiddenNodes, numInputs, numOutputs, 0,cuOutputLayerAct);
			hipDeviceSynchronize();
			


        // Transfer the memory off of the GPU to the CPU
        hipMemcpy(outputLayer, cuOutputLayer, numOutputs * sizeof(float), hipMemcpyDeviceToHost);

        printf("%f\n", outputLayer[0]);

}


